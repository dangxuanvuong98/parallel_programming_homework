
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mykernel(void) {
	printf("Hello World!\n");
}

int main(void) {
	mykernel<<<1, 1>>>();
	printf("Hello World!\n");
	return 0;
}